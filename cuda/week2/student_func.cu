#include "hip/hip_runtime.h"
// Homework 2
// Image Blurring
//****************************************************************************

#include "reference_calc.cpp"
#include "utils.h"

__global__
void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{

    int y = (threadIdx.y + (blockDim.y * blockIdx.y));
    int x = (threadIdx.x + (blockDim.x * blockIdx.x));
        if ((y >= numRows) || (x >= numCols)) { return;}
        float result = 0.f;
      //For every value in the filter around the pixel (c, r)
      for (int filter_r = -filterWidth/2; filter_r <= filterWidth/2; ++filter_r) {
        for (int filter_c = -filterWidth/2; filter_c <= filterWidth/2; ++filter_c) {
          //Find the global image position for this filter position
          //clamp to boundary of the image
          int image_r = min(max(y + filter_r, 0), static_cast<int>(numRows - 1));
          int image_c = min(max(x + filter_c, 0), static_cast<int>(numCols - 1));

          float image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
          float filter_value = filter[(filter_r + filterWidth/2) * filterWidth + filter_c + filterWidth/2];

          result += image_value * filter_value;
        }
      }

      outputChannel[y * numCols + x] = result;
}


//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
    int y = (threadIdx.y + (blockDim.y * blockIdx.y));
    int x = (threadIdx.x + (blockDim.x * blockIdx.x));
    int i = (y * numCols) + x;
    if (i >= numCols * numRows){ return;} else {
        uchar4 rgba = inputImageRGBA[i];    
        redChannel[i]   = rgba.x;
        greenChannel[i] = rgba.y;
        blueChannel[i]  = rgba.z;
    }
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

//These are weird pointers maybe they need to be change to trailing __
unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original 
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

 
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));
  checkCudaErrors(hipMemset(d_filter, 0, sizeof(float) * filterWidth * filterWidth)); //make sure no memory is left laying around
  checkCudaErrors(hipMemcpy(d_filter, h_filter, 
                                sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));



}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
    const dim3 blockSize(numRows,1,1);
    const dim3 gridSize(1,1,1);

    const int blockWidth = 16;
    const dim3 blockSize2(blockWidth,blockWidth,1);
    const dim3 gridSize2(((numCols/blockWidth) + 1),((numRows/blockWidth) + 1),1);
    
  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
  
  //TODO: Launch a kernel for separating the RGBA image into different color channels
  separateChannels<<<gridSize2, blockSize2>>>(d_inputImageRGBA, numRows, numCols,
                                            d_red, d_green, d_blue);
  
  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  //TODO: Call your convolution kernel here 3 times, once for each color channel.
  gaussian_blur<<<gridSize2, blockSize2>>>(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur<<<gridSize2, blockSize2>>>(d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur<<<gridSize2, blockSize2>>>(d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);
  
  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize2, blockSize2>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
